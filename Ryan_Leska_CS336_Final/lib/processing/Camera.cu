#include "hip/hip_runtime.h"

#include "headers/Camera.hpp"

__device__ Camera::Camera(Vec3 lookfrom, Vec3 lookat, Vec3 vup, float vfov, float aspect, float aperture, float focus_dist) { // vfov is top to bottom in degrees
    lens_radius = aperture / 2.0f;
    float theta = vfov*((float)M_PI)/180.0f;
    float half_height = tan(theta/2.0f);
    float half_width = aspect * half_height;
    origin = lookfrom;
    w = (lookfrom - lookat).normalized();
    u = (vup.cross(w)).normalized();
    v = w.cross(u);
    lower_left_corner = origin  - u*half_width*focus_dist - v*half_height*focus_dist - w*focus_dist;
    horizontal = u*2.0f*half_width*focus_dist;
    vertical = v*2.0f*half_height*focus_dist;
}


__device__ Ray Camera::get_ray(float s, float t, hiprandState *state) {
    Vec3 rd = random_in_unit_disk(state) * lens_radius;
    Vec3 offset = (u * rd.x) + (v * rd.y);
    return Ray(origin + offset, lower_left_corner + horizontal*s + vertical*t - origin - offset);
}


__device__ Vec3 Camera::random_in_unit_disk(hiprandState *state) {
    Vec3 p;
    do {
        p = Vec3(hiprand_uniform(state),hiprand_uniform(state),0) * 2.0f - Vec3(1,1,0);
    } while (p.dot(p) >= 1.0f);
    return p;
}