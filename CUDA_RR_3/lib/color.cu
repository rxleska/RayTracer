#include "hip/hip_runtime.h"
#include "headers/color.hpp"

__host__ inline double linear_to_gamma(double linear_component)
{
    if (linear_component > 0)
        return std::sqrt(linear_component);

    return 0;
}

__host__ double clamp(double x, double min, double max) {
    if (x < min) return min;
    if (x > max) return max;
    return x;
}

__host__ void write_color(std::ostream& out, const color& pixel_color) {
    auto r = pixel_color.x();
    auto g = pixel_color.y();
    auto b = pixel_color.z();

    // Apply a linear to gamma transform for gamma 2
    r = linear_to_gamma(r);
    g = linear_to_gamma(g);
    b = linear_to_gamma(b);

    // Translate the [0,1] component values to the byte range [0,255].
    int rbyte = int(255.999 * clamp(r, 0.0, 0.999));
    int gbyte = int(255.999 * clamp(g, 0.0, 0.999));
    int bbyte = int(255.999 * clamp(b, 0.0, 0.999));

    // Write out the pixel color components.
    out << rbyte << ' ' << gbyte << ' ' << bbyte << '\n';
}