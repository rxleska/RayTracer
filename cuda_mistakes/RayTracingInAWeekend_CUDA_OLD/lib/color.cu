#include "hip/hip_runtime.h"
#include "headers/color.hpp"

__host__ inline double linear_to_gamma(double linear_component)
{
    if (linear_component > 0)
        return sqrt(linear_component);
    return 0;
}

__host__ void write_color(std::ostream& out, const color& pixel_color) {
    double r = pixel_color.x();
    double g = pixel_color.y();
    double b = pixel_color.z();

    // Apply a linear to gamma transform for gamma 2
    r = linear_to_gamma(r);
    g = linear_to_gamma(g);
    b = linear_to_gamma(b);

    // Translate the [0,1] component values to the byte range [0,255].
    //TODO find a static way to do this
    interval intensity = interval(0.0, 0.999);
    int rbyte = int(255.999 * intensity.clamp(r));
    int gbyte = int(255.999 * intensity.clamp(g));
    int bbyte = int(255.999 * intensity.clamp(b));

    // Write out the pixel color components.
    out << rbyte << ' ' << gbyte << ' ' << bbyte << '\n';
}