#include "hip/hip_runtime.h"
#include "headers/hittable_list.hpp"

__host__ __device__ hittable_list::hittable_list() {}
__host__ __device__ hittable_list::hittable_list(hittable* object) { add(object); }

__host__ __device__ void hittable_list::clear() { objects.clear(); }

__host__ __device__ void hittable_list::add(hittable* object)
{
    objects.add(object);
}

__host__ __device__ bool hittable_list::hit(const ray &r, interval ray_t, hit_record &rec) const {
    hit_record temp_rec;
    bool hit_anything = false;
    double closest_so_far = ray_t.max;

    // for (const auto &object : objects)
    for(int i = 0; i < objects.size(); i++)
    {
        hittable* object = objects.get(i);
        if (object->hit(r, interval(ray_t.min, closest_so_far), temp_rec))
        {
            hit_anything = true;
            closest_so_far = temp_rec.t;
            rec = temp_rec;
        }
    }

    return hit_anything;
}
