#include "hip/hip_runtime.h"
// hittable
#include "lib/hittable/headers/Hitable.hpp"
#include "lib/hittable/headers/HitRecord.hpp"
#include "lib/hittable/headers/Scene.hpp"
#include "lib/hittable/headers/Sphere.hpp"
// materials
#include "lib/materials/headers/Material.hpp"
#include "lib/materials/headers/Lambertian.hpp"
// processing
#include "lib/processing/headers/Camera.hpp"
#include "lib/processing/headers/Ray.hpp"
#include "lib/processing/headers/Vec3.hpp"

#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>


// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n" << hipGetErrorString(result) << std::endl;
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

// __global__ void rand_init(hiprandState *rand_state) {
//     if (threadIdx.x == 0 && blockIdx.x == 0) {
//         hiprand_init(1984, 0, 0, rand_state);
//     }
// }

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__device__ Vec3 getColor(const Ray &r, Scene **world, hiprandState *local_rand_state) {
    Ray cur_ray = r;
    Vec3 cur_attenuation = Vec3(1.0,1.0,1.0);
    for(int i = 0; i < 50; i++) {
        HitRecord rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            Ray scattered;
            Vec3 attenuation;
            if(rec.mat->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation = cur_attenuation * attenuation;
                cur_ray = scattered;
            }
            else {
                return Vec3(0.0,0.0,0.0);
            }
        }
        else {
            Vec3 unit_direction = (cur_ray.direction).normalized();
            float t = 0.5f*(unit_direction.y + 1.0f);
            Vec3 c = Vec3(1.0, 1.0, 1.0)*(1.0f-t) + Vec3(0.5, 0.7, 1.0)*t;
            return cur_attenuation * c;
        }
    }
    return Vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void free_world(Hitable **d_list, Scene **d_world, Camera **d_camera) {
    for(int i=0; i < 22*22+1+3; i++) {
        delete ((Sphere *)d_list[i])->mat;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    // Original: Each thread gets same seed, a different sequence number, no offset
    // hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
    // BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
    // performance improvement of about 2x!
    // hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
    hiprand_init(1984+pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(uint8_t *fb, int max_x, int max_y, int ns, Camera **cam, Scene **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    Vec3 col(0,0,0);
    for(int s=0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        Ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col = col + getColor(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col = col * (1.0f/float(ns));
    col.x = sqrt(col.x);
    col.y = sqrt(col.y);
    col.z = sqrt(col.z);
    fb[pixel_index*3+0] = uint8_t(int(255.99*col.x));
    fb[pixel_index*3+1] = uint8_t(int(255.99*col.y));
    fb[pixel_index*3+2] = uint8_t(int(255.99*col.z));
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(Hitable **d_list, Scene **d_world, Camera **d_camera, int nx, int ny, hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new Sphere(Vec3(0,-1000.0,-1), 1000,
                               new Lambertian(Vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for(int a = -11; a < 11; a++) {
            for(int b = -11; b < 11; b++) {
                float choose_mat = RND;
                Vec3 center(a+RND,0.2,b+RND);
                d_list[i++] = new Sphere(center, 0.2,new Lambertian(Vec3(RND*RND, RND*RND, RND*RND)));
                // if(choose_mat < 0.8f) {
                //     d_list[i++] = new Sphere(center, 0.2,
                //                              new lambertian(vec3(RND*RND, RND*RND, RND*RND)));
                // }
                // else if(choose_mat < 0.95f) {
                //     d_list[i++] = new Sphere(center, 0.2,
                //                              new metal(vec3(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
                // }
                // else {
                //     d_list[i++] = new Sphere(center, 0.2, new dielectric(1.5));
                // }
            }
        }
        // d_list[i++] = new sphere(vec3(0, 1,0),  1.0, new dielectric(1.5));
        d_list[i++] = new Sphere(Vec3(-4, 1, 0), 1.0, new Lambertian(Vec3(0.4, 0.2, 0.1)));
        // d_list[i++] = new sphere(vec3(4, 1, 0),  1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world  = new Scene(d_list, 22*22+1+3);

        Vec3 lookfrom(13,2,3);
        Vec3 lookat(0,0,0);
        float dist_to_focus = 10.0; (lookfrom-lookat).length();
        float aperture = 0.1;
        *d_camera   = new Camera(lookfrom,
                                 lookat,
                                 Vec3(0,1,0),
                                 30.0,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus);
    }
}


int main() {
    int nx = 1920/2;
    int ny = 1080/2;
    int ns = 25;
    int tx = 20;
    int ty = 12;

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    // size_t fb_size = num_pixels*sizeof(vec3);
    size_t fb_size = num_pixels*sizeof(uint8_t)*3;

    // allocate FB
    // vec3 *fb;
    uint8_t *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1,1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the camera
    Hitable **d_list;
    // int num_hitables = 22*22+1+3;
    int num_hitables = 22*22+1+1;
    checkCudaErrors(hipMalloc((void **)&d_list, num_hitables*sizeof(Hitable *)));
    Scene **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(Scene *)));
    Camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera *)));
    create_world<<<1,1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny,  ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    start = clock();

    //open file
    FILE *f = fopen("image.ppm", "w");
    fprintf(f, "P6 %d %d 255\n", nx, ny);
    uint8_t *fb2 = (uint8_t *)malloc(fb_size*3);
    //direct memory copy
    checkCudaErrors(hipMemcpy(fb2, fb, fb_size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    fwrite(fb2, sizeof(uint8_t), 3*nx*ny, f);
    fclose(f);

    // Output FB as Image
    // std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    // for (int j = ny-1; j >= 0; j--) {
    //     for (int i = 0; i < nx; i++) {
    //         size_t pixel_index = j*nx + i;
    //         int ir = int(fb[pixel_index*3 + 0]);
    //         int ig = int(fb[pixel_index*3 + 1]);
    //         int ib = int(fb[pixel_index*3 + 2]);
    //         std::cout << ir << " " << ig << " " << ib << "\n";
    //     }
    // }
    // Output FB as Image (uint8_t) P6 PPM

    stop = clock();
    timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "output took " << timer_seconds << " seconds.\n";

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list, d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();
}