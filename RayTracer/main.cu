#include "hip/hip_runtime.h"
// hittable
#include "lib/hittable/headers/Hitable.hpp"
#include "lib/hittable/headers/HitRecord.hpp"
#include "lib/hittable/headers/Octree.hpp"
#include "lib/hittable/headers/Sphere.hpp"
#include "lib/hittable/headers/Polygon_T.hpp"
#include "lib/hittable/headers/Octree.hpp"
// materials
#include "lib/materials/headers/Material.hpp"
#include "lib/materials/headers/Lambertian.hpp"
#include "lib/materials/headers/Metal.hpp"
#include "lib/materials/headers/Dielectric.hpp"
#include "lib/materials/headers/Light.hpp"
#include "lib/materials/headers/LambertianBordered.hpp"
#include "lib/materials/headers/Textured.hpp"
#include "lib/materials/headers/Phong.hpp"
#include "lib/materials/headers/PhongLamb.hpp"
// processing
#include "lib/processing/headers/Camera.hpp"
#include "lib/processing/headers/Ray.hpp"
#include "lib/processing/headers/Vec3.hpp"

#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>

#include <vector>

// Define these only in *one* .cc file.
#define TINYGLTF_IMPLEMENTATION
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
// #define TINYGLTF_NOEXCEPTION // optional. disable exception handling.
#include "lib/external/tiny_gltf.h"

#define MAX_OBJECTS 1000

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n" << hipGetErrorString(result) << std::endl;
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

// __global__ void rand_init(hiprandState *rand_state) {
//     if (threadIdx.x == 0 && blockIdx.x == 0) {
//         hiprand_init(1984, 0, 0, rand_state);
//     }
// }

__global__ void rand_init_singleton(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void init_texture(Vec3 *textures, float *texture, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < width * height) {
        textures[idx] = Vec3(texture[idx * 3], texture[idx * 3 + 1], texture[idx * 3 + 2]);
    }
}


__host__ void allocate_texture(const char *filename, Vec3 **textures, int texture_index) {
    int width, height;
    float *im1 = load_texture(filename, width, height);

    // Allocate memory for device image and copy
    float *d_im1;
    checkCudaErrors(hipMalloc((void **)&d_im1, width * height * 3 * sizeof(float)));
    checkCudaErrors(hipMemcpy(d_im1, im1, width * height * 3 * sizeof(float), hipMemcpyHostToDevice));

    // Allocate memory for the texture on device and initialize in parallel
    Vec3 *d_texture;
    checkCudaErrors(hipMalloc((void **)&d_texture, width * height * sizeof(Vec3)));
    init_texture<<<(width * height + 255) / 256, 256>>>(d_texture, d_im1, width, height);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(&textures[texture_index], &d_texture, sizeof(Vec3 *), hipMemcpyHostToDevice));

    free(im1);
}

__global__ void init_mesh(Vec3 *d_mesh, float *mesh, int num_points, float scale) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_points) {
        d_mesh[idx] = Vec3(mesh[idx * 3], mesh[idx * 3 + 1], mesh[idx * 3 + 2]) * scale;
    }
}

__global__ void place_mesh(Vec3 **meshes, int *mesh_lengths, Vec3 * d_mesh, int mesh_index, int mesh_length) {
   if(threadIdx.x == 0 && blockIdx.x == 0){
        meshes[mesh_index] = d_mesh;
        mesh_lengths[mesh_index] = mesh_length;
    }

}


/*
* Load mesh from file
* gltf_position is found in the .gltf file "meshes": [ { "primitives": [ { "attributes": { "POSITION": 0 } } ] } ]
* gltf_bufferView is found in the .gltf file "bufferViews": [ { "buffer": 0, "byteOffset": 0, "byteLength": 0 } ]
*/
__host__ void allocate_mesh(const char *filename, Vec3 **meshes, int mesh_index, int *mesh_length) {

    //use tinygltf to load the mesh
    tinygltf::Model model;
    tinygltf::TinyGLTF loader;
    std::string err;
    std::string warn;

    bool ret = loader.LoadASCIIFromFile(&model, &err, &warn, filename);
    if (!warn.empty()) {
        std::cout << "Warn: " << warn << std::endl;
    }
    if (!err.empty()) {
        std::cerr << "Err: " << err << std::endl;
    }
    if (!ret) {
        std::cerr << "Failed to parse glTF\n";
        return;
    }

    std::vector<float> floatBuffer;

    for (const auto &mesh : model.meshes) {
        for (const auto &primitive : mesh.primitives) {
            // Ensure the primitive is a triangle mesh
            if (primitive.mode != TINYGLTF_MODE_TRIANGLES) {
                continue;
            }

            const tinygltf::Accessor &indexAccessor = model.accessors[primitive.indices];
            const tinygltf::BufferView &indexBufferView = model.bufferViews[indexAccessor.bufferView];
            const tinygltf::Buffer &indexBuffer = model.buffers[indexBufferView.buffer];

            const tinygltf::Accessor &positionAccessor = model.accessors[primitive.attributes.at("POSITION")];
            const tinygltf::BufferView &positionBufferView = model.bufferViews[positionAccessor.bufferView];
            const tinygltf::Buffer &positionBuffer = model.buffers[positionBufferView.buffer];

            // Index data type (unsigned short, unsigned int, etc.)
            const unsigned char *indexData = indexBuffer.data.data() + indexBufferView.byteOffset + indexAccessor.byteOffset;
            size_t indexStride = tinygltf::GetComponentSizeInBytes(indexAccessor.componentType);

            // Position data (vec3)
            const unsigned char *positionData = positionBuffer.data.data() + positionBufferView.byteOffset + positionAccessor.byteOffset;
            size_t positionStride = positionAccessor.ByteStride(positionBufferView);

            // Iterate over indices in groups of 3 (for triangles)
            for (size_t i = 0; i < indexAccessor.count; i += 3) {
                // Get the 3 indices of the triangle
                unsigned int indices[3];
                for (size_t j = 0; j < 3; ++j) {
                    if (indexAccessor.componentType == TINYGLTF_COMPONENT_TYPE_UNSIGNED_SHORT) {
                        indices[j] = ((const unsigned short *)indexData)[i + j];
                    } else if (indexAccessor.componentType == TINYGLTF_COMPONENT_TYPE_UNSIGNED_INT) {
                        indices[j] = ((const unsigned int *)indexData)[i + j];
                    } else if (indexAccessor.componentType == TINYGLTF_COMPONENT_TYPE_UNSIGNED_BYTE) {
                        indices[j] = ((const unsigned char *)indexData)[i + j];
                    }
                }

                // Fetch the positions corresponding to the indices
                for (size_t j = 0; j < 3; ++j) {
                    const float *vertex = (const float *)(positionData + indices[j] * positionStride);
                    floatBuffer.push_back(vertex[0]);
                    floatBuffer.push_back(vertex[1]);
                    floatBuffer.push_back(vertex[2]);
                }
            }
        }
    }

    //output file
    // FILE *f = fopen("mesh.txt", "w");
    // for(int i = 0; i < floatBuffer.size()/3; i++){
    //     fprintf(f, "%f %f %f\n", floatBuffer[i*3], floatBuffer[i*3+1], floatBuffer[i*3+2]);
    // }
    // fclose(f);

    // // //print out all the vertices
    // for(int i = 0; i < floatBuffer.size(); i++){
    //     printf("%f %f %f\n", floatBuffer[i*3], floatBuffer[i*3+1], floatBuffer[i*3+2]);
    // }

    int gltf_accessor = floatBuffer.size() / 3;
    //convert vector to array
    float *floatBufferArray = floatBuffer.data();

    // // Allocate memory for device image and copy
    float *d_mesh_f; 
    checkCudaErrors(hipMalloc((void **)&d_mesh_f, gltf_accessor * 3 * sizeof(float)));
    checkCudaErrors(hipMemcpy(d_mesh_f, floatBufferArray, gltf_accessor * 3 * sizeof(float), hipMemcpyHostToDevice));
    Vec3 *d_mesh;
    checkCudaErrors(hipMalloc((void **)&d_mesh, gltf_accessor * sizeof(Vec3)));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    init_mesh<<<(gltf_accessor + 255) / 256, 256>>>(d_mesh, d_mesh_f, gltf_accessor, 2.0f);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    place_mesh<<<1,1>>>(meshes, mesh_length, d_mesh, mesh_index, gltf_accessor);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
} 


__device__ Vec3 getColor(const Ray &r, Camera **cam, Scene **world, hiprandState *local_rand_state, bool &edge_hit) {
    Ray cur_ray = r;
    Vec3 cur_attenuation = Vec3(1.0,1.0,1.0);
    for(int i = 0; i < (*cam)->bounces; i++) {
        HitRecord rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            Ray scattered;
            Vec3 attenuation;
            int did_scatter = rec.mat->scatter(cur_ray, rec, attenuation, scattered, local_rand_state);
            edge_hit = rec.edge_hit;
            if(did_scatter == 1) {
                cur_attenuation = cur_attenuation * attenuation;
                cur_ray = scattered;
            }
            else if(did_scatter == 2){ //light hit return color
                return cur_attenuation * attenuation;
            }
            else if(did_scatter == 3) { //phong hit return color
                return (*world)->handlePhong(rec, cam) * cur_attenuation;
            }
            else if(did_scatter == 4) { //phong hit return color
                int bcCount = ((PhongLamb*) rec.mat)->bc;
                if(bcCount < i){
                    return (*world)->handlePhongLamb(rec, cam, scattered, local_rand_state, true) * cur_attenuation;
                }
                else{
                    cur_attenuation = cur_attenuation * (*world)->handlePhongLamb(rec, cam, scattered, local_rand_state, false);
                    cur_ray = scattered;
                }
            }
            else {
                return Vec3(0.0,0.0,1.0);
            }
        }
        else {
            float ambient = (*cam)->ambient_light_level;
            Vec3 unit_direction = (cur_ray.direction).normalized();
            float t = 0.5f*(unit_direction.y + 1.0f);
            Vec3 c = Vec3(1.0, 1.0, 1.0)*(1.0f-t) + Vec3(0.5, 0.7, 1.0)*t;
            return cur_attenuation * (c*ambient);
        }
    }
    return Vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void free_world(Hitable **device_object_list, Scene **d_world, Camera **d_camera) {
    for(int i=0; i < (*d_world)->hitable_count; i++) {
        delete ((Sphere *)device_object_list[i])->mat;
        delete device_object_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

__global__ void rand_init_render(int max_x, int max_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    hiprand_init(1984+pixel_index, 0, 0, &rand_state[pixel_index]);
}

__device__ float clamp(float x, float min, float max) {
    if(x < min) return min;
    if(x > max) return max;
    return x;
}

__global__ void render(uint8_t *fb, int max_x, int max_y, int ns, Camera **cam, Scene **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;

    int pixel_index = (max_y - j - 1)*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    Vec3 col(0,0,0);
    bool edge_hit = false;
    bool edge_hit_check = false;
    int samples = (*cam)->samples;
    for(int s=0; s < samples; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        Ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col = col + getColor(r, cam, world, &local_rand_state, edge_hit_check);

        if(!edge_hit && edge_hit_check) {
            edge_hit = true;
            samples = samples * (*cam)->msaa_x;
        }
    }
    rand_state[pixel_index] = local_rand_state;
    col = col / float(samples);
    

    fb[pixel_index*3+0] = uint8_t(int(255.99*clamp(sqrt(col.x), 0.0f, 1.0f)));
    fb[pixel_index*3+1] = uint8_t(int(255.99*clamp(sqrt(col.y), 0.0f, 1.0f)));
    fb[pixel_index*3+2] = uint8_t(int(255.99*clamp(sqrt(col.z), 0.0f, 1.0f)));
}

#define RND (hiprand_uniform(&local_rand_state))


#include "lib/Scenes/TestScene.hpp"
#include "lib/Scenes/RTIAW.hpp"
#include "lib/Scenes/CornellBox.hpp"
#include "lib/Scenes/PhongCornellBox.hpp"
#include "lib/Scenes/PhongMixCornellBox.hpp"
#include "lib/Scenes/CornellRoomOfMirrors.hpp"
#include "lib/Scenes/Billards.hpp"

__global__ void create_world(Hitable **device_object_list, Scene **d_world, Camera **d_camera, int nx, int ny, hiprandState *rand_state, Vec3 **textures, int num_textures, Vec3 ** meshes, int * mesh_lengths, int num_meshes){
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        create_RTIAW_sample(device_object_list, d_world, d_camera, nx, ny, rand_state);
        // create_test_scene(device_object_list, d_world, d_camera, nx, ny, rand_state, textures, num_textures, meshes, mesh_lengths, num_meshes);
        // create_Cornell_Box_Octree(device_object_list, d_world, d_camera, nx, ny, rand_state);
        // create_Cornell_Box_Octree_ROM(device_object_list, d_world, d_camera, nx, ny, rand_state, textures, num_textures, meshes, mesh_lengths, num_meshes);
        // create_Billards_Scene(device_object_list, d_world, d_camera, nx, ny, rand_state, textures, num_textures, meshes, mesh_lengths, num_meshes);
        // create_Phong_Cornell_Box_Octree(device_object_list, d_world, d_camera, nx, ny, rand_state);
        // create_Phong_Mix_Cornell_Box_Octree(device_object_list, d_world, d_camera, nx, ny, rand_state);
    }
}


int main() {
    //increase stack size
    hipDeviceSetLimit(hipLimitStackSize, 4096);
    // int nx = 512*8;
    // int nx = 500*1;
    int nx = 1440;
    // int ny = 512*8;
    // int ny = 500*1;
    int ny = 900;
    int ns = 100;
    // int tx = 20;
    // int ty = 12;
    // int tx = 16;
    // int ty = 10;
    int tx = 512;
    int ty = 1;

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    // size_t fb_size = num_pixels*sizeof(vec3);
    size_t fb_size = num_pixels*sizeof(uint8_t)*3;

    // allocate Frame Buffer (fb)
    uint8_t *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // timing
    clock_t start, stop;
    start = clock();


    // allocate random state for each pixel
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    // initialize random state for Octree generation
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init_singleton<<<1,1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //LOAD IMAGES
    int num_textures = 2;
    Vec3 **textures; // array of texture arrays
    checkCudaErrors(hipMalloc((void **)&textures, num_textures * sizeof(Vec3 *)));

    // Load image
    allocate_texture("imTexts/Monkey.ppm", textures, 0);
    allocate_texture("imTexts/ExampleImage.ppm", textures, 1);


    // LOAD MESHES
    int num_meshes = 2;
    Vec3 **meshes; // array of mesh point arrays
    int *num_points_meshes; // array of number of points in each mesh
    checkCudaErrors(hipMalloc((void **)&meshes, num_meshes * sizeof(Vec3 *)));
    checkCudaErrors(hipMalloc((void **)&num_points_meshes, num_meshes * sizeof(int)));

    allocate_mesh("meshFiles/man/scene.gltf", meshes, 0, num_points_meshes);
    allocate_mesh("meshFiles/knight/scene.gltf", meshes, 1, num_points_meshes);


    // make our world of hitables & the camera
    Hitable **device_object_list;
    // int num_hitables = 22*22+1+3;
    int num_hitables = MAX_OBJECTS;
    checkCudaErrors(hipMalloc((void **)&device_object_list, num_hitables*sizeof(Hitable *)));
    Scene **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(Octree *)));
    Camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera *)));
    create_world<<<1,1>>>(device_object_list, d_world, d_camera, nx, ny, d_rand_state2, textures, num_textures, meshes, num_points_meshes, num_meshes);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


    // print world created 
    printf("World created\n");
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";


    start = clock();
    // Render our buffer
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    rand_init_render<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny,  ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    start = clock();

    //open file
    FILE *f = fopen("image.ppm", "wb");
    fprintf(f, "P6 %d %d 255\n", nx, ny);
    uint8_t *fb2 = (uint8_t *)malloc(fb_size*3);
    //direct memory copy
    checkCudaErrors(hipMemcpy(fb2, fb, fb_size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    fwrite(fb2, sizeof(uint8_t), 3*nx*ny, f);
    fclose(f);

    stop = clock();
    timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "output took " << timer_seconds << " seconds.\n";

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(device_object_list, d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(device_object_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();
}