#include "hip/hip_runtime.h"
#include "headers/Scene.hpp"

#include "headers/Hitable.hpp"
#include "../processing/headers/Ray.hpp"

#include "../materials/headers/Phong.hpp"
#include "../materials/headers/PhongLamb.hpp"

#include <iostream>


__device__ Scene::Scene(){
    hitable_count = 0;
    hitable_capacity = 2;
    // printf("Scene constructor\n");
    // hitables = new Hitable*[hitable_capacity]; // c++ new operator not supported in cuda
    hitables = (Hitable**)malloc(sizeof(Hitable*) * hitable_capacity);

    //point lights
    pointLights = nullptr;
    point_light_count = 0;
}

__host__ void Scene::free_memory(){
    hipFree(hitables);
}

__device__ void Scene::resize(int new_capacity){
    // printf("Resizing\n");
    // Hitable **new_hitables = new Hitable*[new_capacity];
    Hitable **new_hitables = (Hitable**)malloc(sizeof(Hitable*) * new_capacity);
    for (int i = 0; i < hitable_count; i++) {
        new_hitables[i] = hitables[i];
    }
    // delete[] hitables;
    free(hitables);
    hitables = new_hitables;
    hitable_capacity = new_capacity;

}
__device__ void Scene::addHitable(Hitable *hittable){
    if (hitable_count == hitable_capacity) {
        resize(hitable_capacity * 2);
    }
    hitables[hitable_count++] = hittable;
}
__device__ bool Scene::hit(const Ray &ray, float t_min, float t_max, HitRecord &rec) const{
    bool has_hit = false;
    // float closest = t_max;
    for (int i = 0; i < hitable_count; i++) {
        if (hitables[i]->hit(ray, t_min, t_max, rec)) {
            has_hit = true;
            t_max = rec.t;
            // rec = current_hit;
        }
    }

    return has_hit;
}

__device__ Scene::Scene(Hitable **hitables, int hitable_count){
    this->hitable_count = hitable_count;
    this->hitable_capacity = hitable_count;
    this->hitables = (Hitable**)malloc(sizeof(Hitable*) * hitable_count);
    for (int i = 0; i < hitable_count; i++) {
        this->hitables[i] = hitables[i];
    }
}


__device__ void Scene::empty(){
    hitable_count = 0;
}


__device__ void Scene::debug_print() const{
    for(int i = 0; i < hitable_count; i++){
        hitables[i]->debug_print();
    }
}

__device__ void Scene::setPointLights(Vec3 *pointLights, int light_count){
    this->pointLights = (Vec3*)malloc(sizeof(Vec3) * light_count);
    for (int i = 0; i < light_count; i++) {
        this->pointLights[i] = pointLights[i];
    }
    this->point_light_count = light_count;
}


__device__ Vec3 Scene::handlePhong(const HitRecord &rec, Camera **cam) const{
    Phong *material = (Phong*) rec.mat;

    Vec3 returned_color = Vec3(1.0,1.0,1.0) * (*cam)->ambient_light_level * material->kConsts.z;

    // N_hat normal out of the surface
    Vec3 N_hat = rec.normal;
    N_hat.normalize();

    // vector towards the camera
    Vec3 V_hat = (*cam)->origin - rec.p;
    V_hat.normalize();

    for(int i = 0; i < point_light_count; i+=2){
        //vector towards the light
        Vec3 L_hat_m = pointLights[i] - rec.p; 
        L_hat_m.normalize();

        float Lm_dot_N = L_hat_m.dot(N_hat);

        float time = (pointLights[i] - rec.p).x / L_hat_m.x;
        Ray check_ray = Ray(rec.p, L_hat_m);
        HitRecord check_rec; //this is not used but a null pointer could lead to errors or undefined behavior
        if(!hit(check_ray, 0.001, time, check_rec)){
            //kd * Lm_dot_N * imd
            returned_color = returned_color + pointLights[i+1] * (Lm_dot_N * material->kConsts.y);

            Vec3 R_hat = (N_hat * 2.0f * Lm_dot_N ) - L_hat_m;
            R_hat.normalize();
            //ks * (R_hat dot V_hat)^a * ims
            float R_dot_V = R_hat.dot(V_hat);
            if(R_dot_V > 0){
                returned_color = returned_color + pointLights[i+1] * pow(R_dot_V, material->a) * material->kConsts.x;
            }
        }
        
    }


    return returned_color * material->albedo;
}

__device__ Vec3 Scene::handlePhongLamb(const HitRecord &rec, Camera **cam, Ray &scattered, hiprandState *local_rand_state, bool usePhong) const{
    PhongLamb *material = (PhongLamb*) rec.mat;

    if(usePhong){
        Vec3 returned_color = Vec3(1.0,1.0,1.0) * (*cam)->ambient_light_level * material->kConsts.z;

        // N_hat normal out of the surface
        Vec3 N_hat = rec.normal;
        N_hat.normalize();

        // vector towards the camera
        Vec3 V_hat = (*cam)->origin - rec.p;
        V_hat.normalize();

        for(int i = 0; i < point_light_count; i+=2){
            //vector towards the light
            Vec3 L_hat_m = pointLights[i] - rec.p; 
            L_hat_m.normalize();

            float Lm_dot_N = L_hat_m.dot(N_hat);

            float time = (pointLights[i] - rec.p).x / L_hat_m.x;
            Ray check_ray = Ray(rec.p, L_hat_m);
            HitRecord check_rec; //this is not used but a null pointer could lead to errors or undefined behavior
            if(!hit(check_ray, 0.001, time, check_rec)){
                //kd * Lm_dot_N * imd
                returned_color = returned_color + pointLights[i+1] * (Lm_dot_N * material->kConsts.y);

                Vec3 R_hat = (N_hat * 2.0f * Lm_dot_N ) - L_hat_m;
                R_hat.normalize();
                //ks * (R_hat dot V_hat)^a * ims
                float R_dot_V = R_hat.dot(V_hat);
                if(R_dot_V > 0){
                    returned_color = returned_color + pointLights[i+1] * pow(R_dot_V, material->a) * material->kConsts.x;
                }
            }
            
        }


        return returned_color * material->albedo;
    }
    else{
        Vec3 normal = rec.normal; // get the normal of the hit point

        // get a random unit vector
        Vec3 bounceMod = Vec3::random(-10,10,local_rand_state); // 10 is arbitrary, since we are normalizing it later
        bounceMod.make_unit();

        // get the new direction
        Vec3 target = normal + bounceMod;
        
        // degenerate case where the new direction is close to zero
        if (target.isZero()) {
            target = normal;
        }

        //create new ray
        scattered = Ray(rec.p, target);
        return material->albedo;
    }
}