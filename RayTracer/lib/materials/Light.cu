#include "headers/Light.hpp"

__device__ int Light::scatter(const Ray &ray_in, const HitRecord &rec, Vec3 &attenuation, Ray &scattered_out, hiprandState * rand_state) const {

    // Light materials do not scatter rays they absorb them so we return 2, still set teh attenuation to the color of the light

    if(rec.edge_hit){
        attenuation = color * intensity * 0.1;
    }
    else{
        attenuation = color * intensity;
    }

    //TODO make light use the intensity modifier to make the light brighter or dimmer

    return 2;
}