#include "headers/PhongLamb.hpp"

__device__ int PhongLamb::scatter(const Ray &ray_in, const HitRecord &rec, Vec3 &attenuation, Ray &scattered_out, hiprandState * rand_state) const {
    // phong is handled in a phong function in scene, since it needs multiple parts of the scene to calculate the color
    return 4;
}
