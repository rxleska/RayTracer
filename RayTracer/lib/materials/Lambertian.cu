#include "hip/hip_runtime.h"
#include "headers/Lambertian.hpp"

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#ifndef SAMPLING_METHOD
#define SAMPLING_METHOD 6
#endif

__device__ int Lambertian::scatter(const Ray &ray_in, HitRecord &rec, Vec3 &attenuation, Ray &scattered_out, hiprandState * rand_state) const {
    Vec3 normal = rec.normal; // get the normal of the hit point

    // multiple methods for generating random direction
    #if SAMPLING_METHOD == 1
        Vec3 target = Vec3::random(-10,10, rand_state);
        target.make_unit();
        target = target + normal;
        target.make_unit();
        rec.pdf_passValue = 1.0;
    #elif SAMPLING_METHOD == 2
        Vec3 target = Vec3::random_on_hemisphere(rand_state, normal);
        rec.pdf_passValue = 1.0;
    #elif SAMPLING_METHOD == 6
        Vec3 target = Vec3::random_on_hemisphere_cosine_weighted(rand_state, normal, rec.pdf_passValue);
    #elif SAMPLING_METHOD == 3
        Vec3 target = Vec3::random_on_hemisphere_powerweighted_cosine(rand_state, normal, 2, rec.pdf_passValue);
    #elif SAMPLING_METHOD == 4
        Vec3 target = Vec3::random_on_hemisphere_beckmann(rand_state, normal, 2, rec.pdf_passValue);
    #elif SAMPLING_METHOD == 5
        Vec3 target = Vec3::random_on_hemisphere_blinn_phong(rand_state, normal, 2, rec.pdf_passValue);
    #endif

    // degenerate case where the new direction is close to zero
    if (target.isZero()) {
        target = normal;
    }

    //create new ray
    scattered_out = Ray(rec.p, target);

    //set the attenuation (color modification)
    attenuation = albedo;
    return 5;
}
__device__ double Lambertian::importance_pdf(const Ray &ray_in, const HitRecord &rec, const Ray &scattered, Vec3 *lightPoints, int lightCount) const {


    Vec3 dirToLight = Vec3(0,0,0);
    for(int i = 0; i < lightCount; i+=2){
        dirToLight = dirToLight + (lightPoints[i] - rec.p);
    }
    dirToLight.make_unit();


    // the pdf for lambertian is cos(theta) / pi
    // double cosine = rec.normal.dot(scattered.direction);
    double cosine = dirToLight.dot(scattered.direction);
    if (cosine < 0) {
        cosine = 0;
    }
    return cosine / M_PI;    
}
