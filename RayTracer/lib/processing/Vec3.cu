#include "hip/hip_runtime.h"
#include "headers/Vec3.hpp"
#include <cmath>

__device__ void Vec3::normalize() {
    float m = mag();
    x /= m;
    y /= m;
    z /= m;
}

__device__ void Vec3::make_unit() {
    float mag = sqrt(x * x + y * y + z * z);
    x /= mag;
    y /= mag;
    z /= mag;
}

__device__ Vec3 Vec3::random(hiprandState *state) {
    return Vec3(hiprand_uniform(state), hiprand_uniform(state), hiprand_uniform(state));
}

__device__ Vec3 Vec3::random(float min, float max, hiprandState *state) {
    return Vec3(min + (max - min) * hiprand_uniform(state), min + (max - min) * hiprand_uniform(state), min + (max - min) * hiprand_uniform(state));
}

#ifndef M_PI 
#define M_PI 3.14159265358979323846
#endif

__device__ Vec3 Vec3::random_on_hemisphere(hiprandState *state, const Vec3 &normal) {
    float h0 = hiprand_uniform(state);
    float h1 = hiprand_uniform(state);
    float theta = acos(h0); //between 0 and pi/2 since h0 is between 0 and 1
    float phi = 2 * M_PI * h1; // between 0 and 2pi

    //TODO CHECK THAT MY ROTATION OF z oriented hemisphere to normal is correct

    Vec3 hemisphere = Vec3(sin(theta) * cos(phi), sin(theta) * sin(phi), cos(theta));
    
    Vec3 r = normal.cross(Vec3(0,0,1));
    double rcos = normal.dot(Vec3(0,0,1));
    rcos = rcos / (normal.mag());
    double rtheta = -acos(rcos);

    hemisphere = hemisphere * rcos + r * r.dot(hemisphere) * (1 - rcos) + r.cross(hemisphere) * sin(rtheta);

    return hemisphere;
}
