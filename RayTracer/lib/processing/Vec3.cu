#include "hip/hip_runtime.h"
#include "headers/Vec3.hpp"
#include <cmath>

__device__ void Vec3::normalize() {
    float m = mag();
    x /= m;
    y /= m;
    z /= m;
}

__device__ void Vec3::make_unit() {
    float mag = sqrt(x * x + y * y + z * z);
    x /= mag;
    y /= mag;
    z /= mag;
}

__device__ Vec3 Vec3::random(hiprandState *state) {
    return Vec3(hiprand_uniform(state), hiprand_uniform(state), hiprand_uniform(state));
}

__device__ Vec3 Vec3::random(float min, float max, hiprandState *state) {
    return Vec3(min + (max - min) * hiprand_uniform(state), min + (max - min) * hiprand_uniform(state), min + (max - min) * hiprand_uniform(state));
}
